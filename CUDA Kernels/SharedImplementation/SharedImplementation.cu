#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <chrono>
#include <cmath>
#include <ctime>
#include <random>
#include "HelperFunctions.h"
#include "ErrorChecker.cuh"

#define NUMBER_OF_CYCLES 1000
#define CYCLES_PER_IMAGE 2
#define TILE_WIDTH 256
__constant__ int N = 2048; // match it to total number of objects

const unsigned int SEED_VALUE = 2024;
const bool DRY_RUN = false;

hipError_t nbodyHelperFunction(MassObject** allArrs, int* remainingObjs, int px, int py, int stepsize, double& calculationTime);

__device__ float CUDA_GRAV_CONST = 6.67e-11;

__global__ void calculateSharedAcc(float3* pos, float2* out, int size) {
    //x and y are the 2D positions and z is the weight of the particle  
    extern __shared__ float3 shPosition[];
    float3* globalPos = (float3*)pos;

    int i, j, tile;
    float2 acc = { 0.0f, 0.0f };
    int gtid = blockIdx.x * blockDim.x + threadIdx.x;
    
    //operates tile by tile for optimization (may change logic this later on)
    for (i = 0, tile = 0; i < N; i += TILE_WIDTH, tile++)
    {
        int idx = tile * blockDim.x + threadIdx.x;
        shPosition[threadIdx.x] = globalPos[idx];
        __syncthreads();

        //acceleration calculation
        for (j = 0; j < blockDim.x; j++) {
            float2 vec;

            //vector from current particle to its computational partner particle
            vec.x = pos[gtid].x - shPosition[j].x;
            vec.y = pos[gtid].y - shPosition[j].y;
            //distance squared calculation
            float sqrddist = vec.x * vec.x + vec.y * vec.y;

            if (sqrddist > 0) {
                //net_acc  from this object
                float net_acc = -CUDA_GRAV_CONST * pos[j].z / sqrddist;

                //increment acceleration
                acc.x += cosf(atan2f(vec.y, vec.x)) * net_acc;
                acc.y += sinf(atan2f(vec.y, vec.x)) * net_acc;
            }
        }
    }
    out[gtid] = acc;
}

//semi-randomly initialize the MassObjects given the field size and the number of objects
//all objects are randomly initialized with a mass between 10^22 kg to 10^24 kg
//40% of the objects will be initialized in a central 2.5*10^10 by 2.5*10^10 field
//The remaining 60% can spawn anywhere in the frame's field
void init(int px, int pz, int numberOfObjects, MassObject* arr) {
    int benchmark1 = numberOfObjects * 4 / 10;
    for (int i = 0; i < benchmark1; i++) {
        float x = (0.5 + randfloat(0, 2.5)) * (float)pow(10, 10);
        float y = (0.5 + randfloat(0, 2.5)) * (float)pow(10, 10);
        float vx = rand() % (500) - 250.;
        vx *= (float)pow(10, 3);
        float vy = rand() % (500) - 250.;
        vy *= (float)pow(10, 3);
        float mass = (rand() % 100 + 1) * (float)pow(10, 22);
        *(arr + i) = MassObject(x, y, vx, vy, mass, i);
    }
    for (int i = benchmark1; i < numberOfObjects; i++) {
        float x = (randfloat(0, 4)) * (float)pow(10, 10);
        float y = (randfloat(0, 4)) * (float)pow(10, 10);
        float vx = rand() % (500) - 250;
        vx *= (float)pow(10, 4);
        float vy = rand() % (500) - 250;
        vy *= (float)pow(10, 4);
        float mass = (rand() % 100 + 1) * (float)pow(10, 22);
        *(arr + i) = MassObject(x, y, vx, vy, mass, i);
    }
}

//initialize objects in a normal distribution, centered on fieldX/2 and fieldY/2
void init2(float fieldX, float fieldY, int numberOfObjects, MassObject* arr) {
    std::default_random_engine generator;
    generator.seed(SEED_VALUE);
    std::normal_distribution<float> distributionX(fieldX / 2, fieldX / 4);
    std::normal_distribution<float> distributionY(fieldY / 2, fieldY / 4);
    std::normal_distribution<float> distributionV(0, 500);

    for (int i = 0; i < numberOfObjects; i++) {
        float x = distributionX(generator);
        float y = distributionY(generator);
        float vx = distributionV(generator);
        float vy = distributionV(generator);
        float mass = (rand() % 100 + 1) * (float)pow(10, 22);
        *(arr + i) = MassObject(x, y, vx, vy, mass, i);
    }
}

int main()
{
    srand(SEED_VALUE);
    int px = 800;
    int pz = 800;
    int numberOfObjects = 1024;
    float stepsize = 7200;
    std::cout << "The frame width is " << px << "." << std::endl;
    std::cout << "The frame height is " << pz << "." << std::endl;
    std::cout << "The number of objects used is " << numberOfObjects << "." << std::endl;

    //initialize objects
    MassObject** allArrs = new MassObject * [NUMBER_OF_CYCLES];
    allArrs[0] = new MassObject[numberOfObjects];
    int* remainingObjs = new int[NUMBER_OF_CYCLES];
    remainingObjs[0] = numberOfObjects;
    init2(FIELDX, FIELDY, numberOfObjects, allArrs[0]);

    std::cout << "MassObjects initialized" << std::endl;
    std::cout << "Beginning simulation... " << std::endl;
    double calculationTime = 0;
    std::chrono::time_point<std::chrono::system_clock> start, end;
    start = std::chrono::system_clock::now();

    //perform simulations
    nbodyHelperFunction(allArrs, remainingObjs, px, pz, stepsize, calculationTime);

    end = std::chrono::system_clock::now();
    std::chrono::duration<double> elapsed_time = end - start;

    std::cout << "Simulation completed in " << elapsed_time.count() << " s\n";
    std::cout << "Time spent simulating: " << calculationTime << " s\n";

    // write allArrs data into a text file
    std::ofstream myfile;
    myfile.open("objectsData.txt");
    std::cout << "Output data to objectsData.txt...\n";
    for (int i = 0; i < NUMBER_OF_CYCLES; i++) {
        for (int j = 0; j < remainingObjs[i]; j++) {
            myfile << allArrs[i][j].getObjNumber();
            myfile << " " << allArrs[i][j].getMass();

            myfile << " " << allArrs[i][j].getPosition_x();
            myfile << " " << allArrs[i][j].getPosition_y();

            myfile << " " << allArrs[i][j].getax();
            myfile << " " << allArrs[i][j].getay();

            myfile << " " << allArrs[i][j].getvx();
            myfile << " " << allArrs[i][j].getvy() << std::endl;
        }
    }
    myfile.close();

    // draw frames if not a dry run
    if (!DRY_RUN) {
        //initialize output buffer
        unsigned char*** buffer = new unsigned char** [pz];
        for (int i = 0; i < pz; i++)
        {
            buffer[i] = new unsigned char* [px];
            for (int j = 0; j < px; j++)
            {
                buffer[i][j] = new unsigned char[3];
            }
        }

        std::cout << "Buffer initialized and drawing frames..." << std::endl;
        for (int i = 0; i < NUMBER_OF_CYCLES; i += CYCLES_PER_IMAGE) {
            fill_background(buffer, px, pz, BACKGROUND_COLOR);
            for (int j = 0; j < remainingObjs[i]; j++) {
                struct r_circle thisObject;
                set_circle_values(thisObject, allArrs[i][j], px, pz);
                fill_circle(buffer, px, pz, thisObject);
            }
            // write a new img every CYCLES_PER_IMAGE
            write_bmp_file(i / CYCLES_PER_IMAGE, buffer, px, pz);
        }
        std::cout << "Output images generated." << std::endl;
        delete[] buffer;
    }

    for (int i = 0; i < NUMBER_OF_CYCLES; i++) {
        delete(allArrs[i]);
    }
    delete[] allArrs;
    delete[] remainingObjs;
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t nbodyHelperFunction(MassObject** allArrs, int* remainingObjs, int px, int pz, int stepsize, double& calculationTime)
{
    hipError_t cudaStatus;
    std::chrono::time_point<std::chrono::system_clock> start, end;
    std::chrono::duration<double> sumTime = std::chrono::seconds::zero();

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = checkCuda(hipSetDevice(0));
    if (cudaStatus != hipSuccess) {
        fprintf(stdout, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // initialize device pointers
    float3* dev_accIn;
    float2* dev_accOut;
    cudaStatus = checkCuda(hipMalloc((void**)&dev_accIn, remainingObjs[0] * sizeof(float3)));
    if (cudaStatus != hipSuccess) {
        fprintf(stdout, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = checkCuda(hipMalloc((void**)&dev_accOut, remainingObjs[0] * sizeof(float3)));
    if (cudaStatus != hipSuccess) {
        fprintf(stdout, "hipMalloc failed!");
        hipFree(dev_accIn);
        goto Error;
    }

    for (int i = 1; i < NUMBER_OF_CYCLES; i++) {
        // cudaCopy the ax, ay, and mass from objArray to dev_accIn
        float3* accIn = (float3*)malloc(remainingObjs[i - 1] * sizeof(float3));
        for (int j = 0; j < remainingObjs[i - 1]; j++) {
            accIn[j].x = allArrs[i - 1][j].getPosition_x();
            accIn[j].y = allArrs[i - 1][j].getPosition_y();
            accIn[j].z = allArrs[i - 1][j].getMass();
        }

        cudaStatus = checkCuda(hipMemcpy(dev_accIn, accIn, remainingObjs[i - 1] * sizeof(float3), hipMemcpyHostToDevice));
        if (cudaStatus != hipSuccess) {
            fprintf(stdout, "hipMemcpy failed!");
            checkCuda(hipFree(dev_accIn));
            checkCuda(hipFree(dev_accOut));
            goto Error;
        }

        dim3 threadsPerBlock(TILE_WIDTH);
        dim3 blocks(1 + remainingObjs[i - 1] / TILE_WIDTH);
        start = std::chrono::system_clock::now();
        calculateSharedAcc << <threadsPerBlock, blocks >> > (dev_accIn, dev_accOut, remainingObjs[i - 1]);
        end = std::chrono::system_clock::now();
        sumTime += end - start;

        // Check for any errors launching the kernel
        cudaStatus = checkCuda(hipGetLastError());
        if (cudaStatus != hipSuccess) {
            fprintf(stdout, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            checkCuda(hipFree(dev_accIn));
            checkCuda(hipFree(dev_accOut));
            goto Error;
        }

        // call hipDeviceSynchronize() to wait for the kernel to finish, and return
        // any errors encountered during the launch.
        cudaStatus = checkCuda(hipDeviceSynchronize());
        if (cudaStatus != hipSuccess) {
            fprintf(stdout, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            checkCuda(hipFree(dev_accIn));
            checkCuda(hipFree(dev_accOut));
            goto Error;
        }

        // retrieve result data from device back to host
        float2* accOut = (float2*)malloc(remainingObjs[i - 1] * sizeof(float2));
        cudaStatus = checkCuda(hipMemcpy(accOut, dev_accOut, remainingObjs[i - 1] * sizeof(float2), hipMemcpyDeviceToHost));
        if (cudaStatus != hipSuccess) {
            fprintf(stdout, "hipMemcpy failed!");
            checkCuda(hipFree(dev_accIn));
            checkCuda(hipFree(dev_accOut));
            free(accIn);
            free(accOut);
            goto Error;
        }

        // Update allArrs with the result velocity and positions
        // init current iteration
        allArrs[i] = (MassObject*)malloc(remainingObjs[i - 1] * sizeof(MassObject));
        for (int j = 0; j < remainingObjs[i - 1]; j++) {
            MassObject currentObj = allArrs[i - 1][j];
            currentObj.setAcceleration(accOut[j].x, accOut[j].y);
            currentObj.changePosition(stepsize);
            allArrs[i][j] = currentObj;
            /*allArrs[i][j].setAcceleration(accOut[j].x, accOut[j].y);
            allArrs[i][j].changePosition(stepsize);*/
        }

        // Check for collisions and update arr contents
        //check if any objects have collided
        remainingObjs[i] = check_collisions(allArrs[i], remainingObjs[i - 1], px, pz);

        free(accIn);
        free(accOut);
    }

    // hipDeviceReset( ) must be called in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = checkCuda(hipDeviceReset());
    if (cudaStatus != hipSuccess) {
        fprintf(stdout, "hipDeviceReset failed!");
        checkCuda(hipFree(dev_accIn));
        checkCuda(hipFree(dev_accOut));
        goto Error;
    }

Error:
    calculationTime = sumTime.count();

    return cudaStatus;
}
